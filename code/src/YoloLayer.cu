#include "hip/hip_runtime.h"
#include "YoloConfigs.h"
#include "YoloLayer.h"

using namespace Yolo;

namespace nvinfer1
{
    YoloLayerPlugin::YoloLayerPlugin(const int cudaThread /*= 512*/):mThreadCount(cudaThread)
    {
        mClassCount = CLASS_NUM;
        mYoloKernel.clear();
        mYoloKernel.push_back(yolo1);
        mYoloKernel.push_back(yolo2);
        mYoloKernel.push_back(yolo3);

        mKernelCount = mYoloKernel.size();
    }
    
    YoloLayerPlugin::~YoloLayerPlugin()
    {
        if(mInputBuffer)
            CUDA_CHECK(hipHostFree(mInputBuffer));

        if(mOutputBuffer)
            CUDA_CHECK(hipHostFree(mOutputBuffer));
    }
    
    // create the plugin at runtime from a byte stream
    YoloLayerPlugin::YoloLayerPlugin(const void* data, size_t length)
    {
        using namespace Tn;
        const char *d = reinterpret_cast<const char *>(data), *a = d;
        read(d, mClassCount);
        read(d, mThreadCount);
        read(d, mKernelCount);
        mYoloKernel.resize(mKernelCount);
        auto kernelSize = mKernelCount*sizeof(YoloKernel);
        memcpy(mYoloKernel.data(),d,kernelSize);
        d += kernelSize;

        assert(d == a + length);
    }

    void YoloLayerPlugin::serialize(void* buffer)
    {
        using namespace Tn;
        char* d = static_cast<char*>(buffer), *a = d;
        write(d, mClassCount);
        write(d, mThreadCount);
        write(d, mKernelCount);
        auto kernelSize = mKernelCount*sizeof(YoloKernel);
        memcpy(d,mYoloKernel.data(),kernelSize);
        d += kernelSize;

        assert(d == a + getSerializationSize());
    }
    
    size_t YoloLayerPlugin::getSerializationSize()
    {  
        return sizeof(mClassCount) + sizeof(mThreadCount) + sizeof(mKernelCount) + sizeof(Yolo::YoloKernel) * mYoloKernel.size();
    }

    int YoloLayerPlugin::initialize()
    { 
            int totalCount = 0;
            for(const auto& yolo : mYoloKernel)
                totalCount += (LOCATIONS + 1 + mClassCount) * yolo.width*yolo.height * CHECK_COUNT;
            CUDA_CHECK(hipHostAlloc(&mInputBuffer, totalCount * sizeof(float), hipHostMallocDefault));

            totalCount = 0;//detection count
            for(const auto& yolo : mYoloKernel)
                totalCount += yolo.width*yolo.height * CHECK_COUNT;
            CUDA_CHECK(hipHostAlloc(&mOutputBuffer, sizeof(float) + totalCount * sizeof(Detection), hipHostMallocDefault));
            return 0;
    }
    
    Dims YoloLayerPlugin::getOutputDimensions(int index, const Dims* inputs, int nbInputDims)
    {
            //output the result to channel
            int totalCount = 0;
            for(const auto& yolo : mYoloKernel)
                totalCount += yolo.width*yolo.height * CHECK_COUNT * sizeof(Detection) / sizeof(float);

            return Dims3(totalCount + 1, 1, 1);
    }

    void YoloLayerPlugin::forwardCpu(const float*const * inputs, float* outputs, hipStream_t stream)
    {
            auto Logist = [=](float data){
                return 1./(1. + exp(-data));
            };

            CUDA_CHECK(hipStreamSynchronize(stream));
            int i = 0;
            float* inputData = (float *)mInputBuffer; 
            for(const auto& yolo : mYoloKernel)
            {
                int size = (LOCATIONS + 1 + mClassCount) * yolo.width*yolo.height * CHECK_COUNT;
                CUDA_CHECK(hipMemcpyAsync(inputData, inputs[i], size * sizeof(float), hipMemcpyDeviceToHost, stream));
                inputData += size;
                ++ i;
            }

            inputData = (float *)mInputBuffer;
            std::vector <Detection> result;
            for (const auto& yolo : mYoloKernel)
            {
                int stride = yolo.width*yolo.height;
                for (int j = 0;j < stride ;++j)
                {
                    for (int k = 0;k < CHECK_COUNT; ++k )
                    {
                        int beginIdx = (LOCATIONS + 1 + mClassCount)* stride *k + j;
                        int objIndex = beginIdx + LOCATIONS*stride;
                        
                        //check obj
                        float objProb = Logist(inputData[objIndex]);   
                        if(objProb <= IGNORE_THRESH)
                            continue;

                        Detection det;
                        //det.objectness = objProb;
                        int row = j / yolo.width;
                        int cols = j % yolo.width;

                        //Location
                        det.bbox[0] = (cols + Logist(inputData[beginIdx]))/ yolo.width;
                        det.bbox[1] = (row + Logist(inputData[beginIdx+stride]))/ yolo.height;
                        det.bbox[2] = exp(inputData[beginIdx+2*stride]) * yolo.anchors[2*k];
                        det.bbox[3] = exp(inputData[beginIdx+3*stride]) * yolo.anchors[2*k + 1];

                        //classes
                        std::vector<float> classProb;
                        classProb.resize(mClassCount);
                        for (int c = 0;c<mClassCount;++c)
                        {
                            float cProb =  Logist(inputData[beginIdx + (5+c)*stride ]) * objProb;
                            classProb[c] = (cProb > IGNORE_THRESH) ? cProb : 0;
                        }

                        auto maxEle = std::max_element(&classProb[0], &classProb[mClassCount] + 1);
                        det.classId = std::distance(&classProb[0],maxEle);
                        det.prob = *maxEle;

                        result.emplace_back(det);
                    }
                }

                inputData += (LOCATIONS + 1 + mClassCount) * stride * CHECK_COUNT;
            }

            
            int detCount =result.size();
            auto data = (float *)mOutputBuffer;
            //copy count;
            data[0] = (float)detCount;
            //std::cout << "detCount"<< detCount << std::endl;
            data++;
            //copy result
            memcpy(data,result.data(),result.size()*sizeof(Detection));

            std::cout << "copy result" << std::endl;
            //(count + det result)
            CUDA_CHECK(hipMemcpyAsync(outputs, mOutputBuffer, sizeof(float) + result.size()*sizeof(Detection), hipMemcpyHostToDevice, stream));
    };

    __device__ float Logist(float data){ return 1./(1. + exp(-data)); };

    __global__ void CalDetection(const float *input, float *output,int noElements, 
            int yoloWidth,int yoloHeight,const float anchors[CHECK_COUNT*2],int classes) {
 
        int idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= noElements) return;

        int stride = yoloWidth*yoloHeight;

        for (int k = 0;k < CHECK_COUNT; ++k )
        {
            int beginIdx = (LOCATIONS + 1 + classes)* stride *k + idx;
            int objIndex = beginIdx + LOCATIONS*stride;
            
            //check obj
            float objProb = Logist(input[objIndex]);   
            if(objProb <= IGNORE_THRESH)
                continue;

            //add first result count
            int resCount = (int)atomicAdd(output,1);
            char* data = (char * )output + sizeof(float) + resCount*sizeof(Detection);

            int row = idx / yoloWidth;
            int cols = idx % yoloWidth;

            Detection* det =  (Detection*)(data);
            //det->objectness = objProb;

            //Location
            det->bbox[0] = (cols + Logist(input[beginIdx]))/ yoloWidth;
            det->bbox[1] = (row + Logist(input[beginIdx+stride]))/ yoloHeight;
            det->bbox[2] = exp(input[beginIdx+2*stride]) * anchors[2*k];
            det->bbox[3] = exp(input[beginIdx+3*stride]) * anchors[2*k + 1];
            
            //classes
            float max = IGNORE_THRESH;
            for (int c = 0;c<classes;++c){
                float cProb =  Logist(input[beginIdx + (5 + c) * stride]) * objProb;
                if(cProb > max){
                    max = cProb;
                    det->classId = c;
                    det->prob = max;
                }
            }
        }
    }
   
    void YoloLayerPlugin::forwardGpu(const float *const * inputs,float * output,hipStream_t stream) {
        int numElem;
        void* devAnchor;
        size_t AnchorLen = sizeof(float)* CHECK_COUNT*2;
        CUDA_CHECK(hipMalloc(&devAnchor,AnchorLen));

        //first detect count init 0
        CUDA_CHECK(hipMemset(output, 0, sizeof(float)));
        for (int i = 0;i< mYoloKernel.size();++i)
        {
            const auto& yolo = mYoloKernel[i];
            numElem = yolo.width*yolo.height;

            //copy anchor to device
	        CUDA_CHECK(hipMemcpy(devAnchor,yolo.anchors,AnchorLen,hipMemcpyHostToDevice));

            CalDetection<<< (yolo.width*yolo.height + mThreadCount - 1) / mThreadCount, mThreadCount>>>
                    (inputs[i],output, numElem, yolo.width, yolo.height, (float *)devAnchor, mClassCount);
        }
        CUDA_CHECK(hipFree(devAnchor));
    }


    int YoloLayerPlugin::enqueue(int batchSize, const void*const * inputs, void** outputs, void* workspace, hipStream_t stream)
    {
        assert(batchSize == 1);
        
        //GPU
        forwardGpu((const float *const *)inputs,(float *)outputs[0],stream);

        //CPU
        //forwardCpu((const float *const *)inputs,(float *)outputs[0],stream);
        return 0;
    };

}
